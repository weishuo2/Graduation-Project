#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <thrust/count.h>
#include <numeric>
#include <random>
#include <iostream>
#include <algorithm>
#include <iterator>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_runtime_api.h>
#include "coloring.h"
#include "time.h"
#include "utility.h"

#define MAX_GRID_SIZE 65535

int *same_color_sum;

inline int num_finished(const int n_vertices, 
                 const int * row_ptr, 
                 const int * col, 
                 const int * col_ptr, 
                 const int * row, 
                 const int * colors)
{
     int count = 0;
     int finished;
     for (int i = 0; i < n_vertices; ++i)
     {
         finished = 1;
         int ic = colors[i];
         for (int j = row_ptr[i]; j < row_ptr[i+1]; ++j)
         {
             if (ic == colors[col[j]]) 
             {
                 finished = 0;
                 break;
             }
         }
         if (finished == 0) continue;

         for (int j = col_ptr[i]; j < col_ptr[i+1]; ++j)
         {
             if (ic == colors[row[j]]) 
             {
                 finished = 0;
                 break;
             }
         }
         if (finished == 0) continue;
         count++;
     }
     return count;
}

inline int num_finished(const int n_vertices, 
                 const int * row_ptr, 
                 const int * col, 
                 const int * col_ptr, 
                 const int * row, 
                 const int * colors,
                 const int * reordering)
{
     int count = 0;
     int finished;
     for (int i = 0; i < n_vertices; ++i)
     {
         finished = 1;
         int ic = colors[reordering[i]];
         for (int j = row_ptr[i]; j < row_ptr[i+1]; ++j)
         {
             if (ic == colors[reordering[col[j]]]) 
             {
                 finished = 0;
                 break;
             }
         }
         if (finished == 0) continue;

         for (int j = col_ptr[i]; j < col_ptr[i+1]; ++j)
         {
             if (ic == colors[reordering[row[j]]]) 
             {
                 finished = 0;
                 break;
             }
         }
         if (finished == 0) continue;
         count++;
     }
     return count;
}

inline int compare(const int n, const int * array1, const int * array2)
{
	int count = 0;
	for(int i = 0; i < n; ++i)
        {
		if (array1[i] == array2[i])
			count++;
	}
	return count;
}

// check if all vertices are colored
__global__ void isAllColoredKernel(const int n, const int * colors, int * ret)
{
    for (int i = 0; i < n; ++i)
    {
        if (colors[i] == -1)
        {
            *ret = 0;
            return;
        }
    }
    *ret = 1;
}


__global__ void jplColorKernel(const int n, 
                                 const int c, 
                                 const int *row_ptr, 
                                 const int *col,
                                 const int *col_ptr, 
                                 const int *row,
                                 const int *randoms, 
                                 int *colors)
{   
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; 
       i < n; 
       i += blockDim.x * gridDim.x) 
  {   
    bool f=true; // true iff you have max random

    // ignore nodes colored earlier
    if ((colors[i] != -1)) continue; 

    int ir = randoms[i];

    // look at neighbors to check their random number
    for (int k = row_ptr[i]; k < row_ptr[i+1]; k++) {        
      // ignore nodes colored earlier (and yourself)
      int j = col[k];
      int jc = colors[j];
      if (((jc != -1) && (jc != c)) || (i == j)) continue; 
      int jr = randoms[j];
      if (ir <= jr) f=false;         
    }

    for (int k = col_ptr[i]; k < col_ptr[i+1]; k++) {        
      // ignore nodes colored earlier (and yourself)
      int j = row[k];
      int jc = colors[j];
      if (((jc != -1) && (jc != c)) || (i == j)) continue; 
      int jr = randoms[j];
      if (ir <= jr) f=false;         
    }

    // assign color if you have the maximum random number
    if (f) colors[i] = c;
  }
}

void jplColor(const int n_vertices, 
              const int n_edges, 
              const int *row_ptr, 
              const int *col, 
              const int * col_ptr, 
              const int * row, 
              int *colors) 
{
    int *randoms; // allocate and init random array 
    randoms = (int*)malloc(sizeof(int) * n_vertices);
    std::iota(randoms, randoms + n_vertices, 0);
    std::random_device rd;
    std::mt19937 g(rd());
    std::shuffle(randoms, randoms + n_vertices, g);
    //std::copy(randoms, randoms + n_vertices, std::ostream_iterator<int>(std::cout, " "));
    //std::cout << "\n";

    thrust::fill(colors, colors + n_vertices, -1); // init colors to -1

    int * dev_randoms;
    int * dev_row_ptr;
    int * dev_col;
    int * dev_col_ptr;
    int * dev_row;
    int * dev_colors;
    hipMalloc(&dev_randoms, sizeof(int) * n_vertices);
    hipMalloc(&dev_row_ptr, sizeof(int) * (n_vertices + 1));
    hipMalloc(&dev_col, sizeof(int) * n_edges);
    hipMalloc(&dev_col_ptr, sizeof(int) * (n_vertices + 1));
    hipMalloc(&dev_row, sizeof(int) * n_edges);
    hipMalloc(&dev_colors, sizeof(int) * n_vertices);
    hipMemcpy(dev_randoms, randoms, sizeof(int) * n_vertices, hipMemcpyHostToDevice);
    hipMemcpy(dev_row_ptr, row_ptr, sizeof(int) * (n_vertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(dev_col, col, sizeof(int) * n_edges, hipMemcpyHostToDevice);
    hipMemcpy(dev_col_ptr, col_ptr, sizeof(int) * (n_vertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(dev_row, row, sizeof(int) * n_edges, hipMemcpyHostToDevice);
    hipMemcpy(dev_colors, colors, sizeof(int) * n_vertices, hipMemcpyHostToDevice);
    
    int is_done;
    int * dev_is_done;
    hipMalloc(&dev_is_done, sizeof(int));
    struct timeval start_time, end_time;
    gettimeofday(&start_time, NULL);
    int iter = 0;
    for(int c=0; c < n_vertices; c++) {
        int nt = 256;
        //int nb = min((n_vertices + nt - 1)/nt, CUDA_MAX_BLOCKS);
        int nb = (n_vertices + nt - 1)/nt; 
        jplColorKernel<<<nb, nt>>>(n_vertices, c, 
                                    dev_row_ptr, dev_col,
                                    dev_col_ptr, dev_row,
                                    dev_randoms, 
                                    dev_colors);

        //isAllColoredKernel<<<1, 1>>>(n_vertices, dev_colors, dev_is_done);
        //hipMemcpy(&is_done, dev_is_done, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(colors, dev_colors, sizeof(int) * n_vertices, hipMemcpyDeviceToHost);
        int num_undone = std::count(colors, colors + n_vertices, -1);
        std::cout << "Iter: " << iter << " "
                  << "num_undone = " << num_undone << " " 
                  << "time = " << time << "ms" << std::endl;
        iter++;
        //if (is_done == 1) break;
    }
    gettimeofday(&end_time, NULL);
    std::cout << "Main loop time: " << elapsed(start_time, end_time) << std::endl;
    hipMemcpy(colors, dev_colors, sizeof(int) * n_vertices, hipMemcpyDeviceToHost);
    hipFree(dev_is_done);
    hipFree(dev_randoms);
    hipFree(dev_row_ptr);
    hipFree(dev_col);
    hipFree(dev_col_ptr);
    hipFree(dev_row);
    hipFree(dev_colors);
    free(randoms);
}

void greedyColor(const int n_vertices, 
                 const int n_edges, 
                 const int *row_ptr, 
                 const int *col, 
                 const int *col_ptr, 
                 const int *row, 
                 int *colors) 
{
  int num_colors = 0; 
  bool * color_used = (bool*)malloc(sizeof(bool) * n_vertices);
  memset(color_used, 0, sizeof(bool) * n_vertices);
  thrust::fill(colors, colors + n_vertices, -1); // init colors to -1

  struct timeval start_time, end_time;

  gettimeofday(&start_time, NULL);
    
     for (int i = 0; i < n_vertices; ++i)
  {
    memset(color_used, 0, sizeof(bool) * num_colors);
    // Mark the colors used by neighbors
    // Traverse its destinations
    for (int j = row_ptr[i]; j < row_ptr[i+1]; ++j)
    {
      int c = colors[col[j]];
      if (c != -1)
        color_used[c] = true;
    }

    // Traverse its sources 
    for (int j = col_ptr[i]; j < col_ptr[i+1]; ++j)
    {
      int c = colors[row[j]];
      if (c != -1)
        color_used[c] = true;
    }

    // select a color which can be used
    for (int j = 0; j < num_colors; ++j)
    {
      if (color_used[j] == false)
        colors[i] = j;
    }
    // add a new color if there is no suitable color
    if (colors[i] == -1)
    {
      colors[i] = num_colors++;
    }
  }

  gettimeofday(&end_time, NULL);
    std::cout << "CPU: Greeddy coloring time in function: " << elapsed(start_time, end_time) << "ms" << std::endl;
  
}

__global__ void checkAllColorsKernel(const int n, 
                             const int *row_ptr, 
                             const int *col,
                             const int *col_ptr, 
                             const int *row, 
                             const int *colors, 
                             int * finished)
{
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; 
       i < n; i += blockDim.x * gridDim.x) 
  {
    int ic = colors[i];
    // Traverse its destinations
    for (int j = row_ptr[i]; j < row_ptr[i+1]; ++j)
    {
      int c = colors[col[j]];
      if (c == ic)
      {
          *finished = 0;
          return;
      }
    }

    // Traverse its sources 
    for (int j = col_ptr[i]; j < col_ptr[i+1]; ++j)
    {
      int c = colors[row[j]];
      if (c == ic)
      {
          *finished = 0;
          return;
      }
    }
  }
}

__global__ void color_kernel(const int n, 
                             const int new_color,
                             const int *row_ptr, 
                             const int *col,
                             const int *col_ptr, 
                             const int *row,
                             const int *pre_colors, 
                             int * ret_colors, 
                             int * continue_flag)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; 
        i < n; i += blockDim.x * gridDim.x) 
    {
        int ic = pre_colors[i];
        int update = 0;
        // Traverse its destinations
        for (int j = row_ptr[i]; j < row_ptr[i+1]; ++j)
        {
          int c = pre_colors[col[j]];
          if (c == ic && col[j] != i)
          {
              update = 1;
              break;
          }
        }

        if (update)
        {
            ret_colors[i] = new_color;
            *continue_flag = 1;
        }
        else
        {
            ret_colors[i] = pre_colors[i];
        }
    }
}

void colorOnGPU(const int n_vertices, 
                const int n_edges, 
                const int *row_ptr, 
                const int *col, 
                const int * col_ptr, 
                const int * row, 
                int *colors) 
{
    //thrust::fill(colors, colors + n_vertices, -1); // init colors to -1
    int init_num_colors = 100;
    for (int i = 0; i < n_vertices; ++i)
       colors[i] = rand() % init_num_colors;

    int * colors2 = (int*)malloc(sizeof(int) * n_vertices);

    int * dev_row_ptr;
    int * dev_col;
    int * dev_col_ptr;
    int * dev_row;
    int * dev_colors;
    int * dev_colors2;
    int * dev_continue_flag;
    hipMalloc(&dev_row_ptr, sizeof(int) * (n_vertices + 1));
    hipMalloc(&dev_col, sizeof(int) * n_edges);
    hipMalloc(&dev_col_ptr, sizeof(int) * (n_vertices + 1));
    hipMalloc(&dev_row, sizeof(int) * n_edges);
    hipMalloc(&dev_colors, sizeof(int) * n_vertices);
    hipMalloc(&dev_colors2, sizeof(int) * n_vertices);
    hipMalloc(&dev_continue_flag, sizeof(int));
    hipMemcpy(dev_row_ptr, row_ptr, sizeof(int) * (n_vertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(dev_col, col, sizeof(int) * n_edges, hipMemcpyHostToDevice);
    hipMemcpy(dev_col_ptr, col_ptr, sizeof(int) * (n_vertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(dev_row, row, sizeof(int) * n_edges, hipMemcpyHostToDevice);
    hipMemcpy(dev_colors, colors, sizeof(int) * n_vertices, hipMemcpyHostToDevice);
    
    int continue_flag = 1;
    struct timeval start_time, end_time;
    gettimeofday(&start_time, NULL);
    int turn_f = 0;
    int new_c = init_num_colors;
    while(continue_flag) {
        turn_f = turn_f ? 0 : 1;
        hipMemset(dev_continue_flag, 0, sizeof(int));
        int nt = 256;
        int nb = (n_vertices + nt - 1)/nt; 
        color_kernel<<<nb, nt>>>(n_vertices, new_c,
                                    dev_row_ptr, dev_col,
                                    dev_col_ptr, dev_row,
                                    (turn_f? dev_colors: dev_colors2), 
                                    (turn_f? dev_colors2: dev_colors),
                                    dev_continue_flag);
        hipMemcpy(&continue_flag, dev_continue_flag, sizeof(int), hipMemcpyDeviceToHost);
        new_c++;
        if (new_c % 100 == 0) {
            std::cout << "num_colors=" << new_c << ", ";
            hipMemcpy(colors, (turn_f?dev_colors2: dev_colors), sizeof(int) * n_vertices, hipMemcpyDeviceToHost);
            hipMemcpy(colors, dev_colors, sizeof(int) * n_vertices, hipMemcpyDeviceToHost);
            hipMemcpy(colors2, dev_colors2, sizeof(int) * n_vertices, hipMemcpyDeviceToHost);
            std::cout << "num_finshed=" << num_finished(n_vertices, row_ptr, col, col_ptr, row, colors) << ", "
                      << "compare=" << compare(n_vertices, colors, colors2) << std::endl;
        }
    }
    gettimeofday(&end_time, NULL);
    std::cout << "Main loop time: " << elapsed(start_time, end_time) << std::endl;
    hipMemcpy(colors, dev_colors, sizeof(int) * n_vertices, hipMemcpyDeviceToHost);
    hipFree(dev_continue_flag);
    hipFree(dev_row_ptr);
    hipFree(dev_col);
    hipFree(dev_col_ptr);
    hipFree(dev_row);
    hipFree(dev_colors);
    hipFree(dev_colors2);
    free(colors2);
}

void colorBycuSPARSE(const int n_vertices, 
                     const int n_edges, 
                     const int *row_ptr, 
                     const int *col, 
                     const int * col_ptr, 
                     const int * row, 
                     int *colors) 
{
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    checkCudaErrors(hipsparseCreate(&handle));
    checkCudaErrors(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseColorInfo_t info;
    checkCudaErrors(hipsparseCreateColorInfo(&info));
    
    float * val = (float*)malloc(sizeof(float) * n_edges);
    for (int i = 0; i < n_edges; ++i)
        val[i] = 1;

    int *d_row_ptr, *d_col;
    float *d_val;
    int *d_colors;
    hipMalloc(&d_colors, sizeof(int) * n_vertices);
    hipMalloc(&d_row_ptr, sizeof(int) * (n_vertices + 1));
    hipMalloc(&d_col, sizeof(int) * n_edges);
    hipMalloc(&d_val, sizeof(float) * n_edges);

    hipMemcpy(d_row_ptr, row_ptr, sizeof(int) * (n_vertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_col, col, sizeof(int) * n_edges, hipMemcpyHostToDevice);
    hipMemcpy(d_val, val, sizeof(float) * n_edges, hipMemcpyHostToDevice);

    float fraction = 1;
    int ncolors = 0;

    int * reordering;
    int * d_reordering;
    reordering = (int*)malloc(sizeof(int) * n_vertices);
    hipMalloc(&d_reordering, sizeof(int) * n_vertices);
    
    checkCudaErrors(hipsparseScsrcolor(handle, n_vertices, n_edges, descr, d_val, d_row_ptr, d_col, &fraction, &ncolors, d_colors, d_reordering, info));
    hipMemcpy(colors, d_colors, sizeof(int) * n_vertices, hipMemcpyDeviceToHost);
    hipMemcpy(reordering, d_reordering, sizeof(int) * n_vertices, hipMemcpyDeviceToHost);
    std::cout << "ncolors=" << ncolors << std::endl;
    std::cout << "1 = " << reordering[1] << ", 2 = " << reordering[403098] << std::endl;
    std::cout << "color: 1 = " << colors[reordering[1]] << ", 2 = " << colors[reordering[403098]] << std::endl;
    int n_finished = num_finished(n_vertices, row_ptr, col, col_ptr, row, colors, reordering);
    std::cout << n_finished << " nodes are correct, but " << n_vertices - n_finished << " are incorrect.\n";
    hipFree(d_row_ptr);
    hipFree(d_col);
    hipFree(d_val);
    hipFree(d_colors);
}

__device__ void acquire_semaphore(volatile int *lock)
{
  while (atomicCAS((int *)lock, 0, 1) != 0);
}

__device__ void release_semaphore(volatile int *lock)
{
  *lock = 0;
  __threadfence();
}

__global__ void colorByEdgeKernel(const int n_edges, 
                                     const int * srcs, 
                                     const int * dsts, 
                                     int * locks, 
                                     int * colors,
                                     int * continue_flag)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; 
        //i < n_edges; i += blockDim.x * blockDim.y * blockDim.z * gridDim.x)
        i < n_edges; i += blockDim.x * gridDim.x)
    {
        int src = srcs[i];
        int dst = dsts[i];
        //acquire_semaphore(&(locks[src]));
        //acquire_semaphore(&(locks[dst]));
        if (colors[src] == colors[dst])
        {
            colors[dst] = colors[src] + 1;
            *continue_flag = 1;
        }
        //release_semaphore(&(locks[src]));
        //release_semaphore(&(locks[dst]));
    }
}

void colorByEdgeOnGPU(const int n_vertices,
                      const int n_edges, 
                      const int * srcs, 
                      const int * dsts, 
                      const int *row_ptr, 
                      const int *col, 
                      const int *col_ptr, 
                      const int *row,
                      int * colors)
{
    int * dev_srcs;
    int * dev_dsts;
    int * dev_colors;
    int * dev_locks;
    int * dev_continue_flag;
    int * locks = (int *)malloc(sizeof(int) * n_vertices);
    memset(colors, 0, sizeof(int) * n_vertices);
    memset(locks, 0, sizeof(int) * n_vertices);
    hipMalloc(&dev_continue_flag, sizeof(int));
    hipMalloc(&dev_srcs, sizeof(int) * n_edges);
    hipMalloc(&dev_dsts, sizeof(int) * n_edges);
    hipMalloc(&dev_colors, sizeof(int) * n_vertices);
    hipMalloc(&dev_locks, sizeof(int) * n_vertices);
    hipMemcpy(dev_srcs, srcs, sizeof(int) * n_edges, hipMemcpyHostToDevice);
    hipMemcpy(dev_dsts, dsts, sizeof(int) * n_edges, hipMemcpyHostToDevice);
    hipMemcpy(dev_colors, colors, sizeof(int) * n_vertices, hipMemcpyHostToDevice);
    hipMemcpy(dev_locks, locks, sizeof(int) * n_vertices, hipMemcpyHostToDevice);
    
    int continue_flag = 1;

    struct timeval start_time, end_time;
    gettimeofday(&start_time, NULL);
    while(continue_flag) {
        int nt = 256;
        int nb = (n_vertices + nt - 1)/nt; 
        nb = nb > MAX_GRID_SIZE ? MAX_GRID_SIZE : nb;
        hipMemset(dev_continue_flag, 0, sizeof(int));
        colorByEdgeKernel<<<nb, nt>>>(n_edges,
                                         dev_srcs, dev_dsts, 
                                         dev_locks, dev_colors,
                                         dev_continue_flag);

        hipError_t err = hipGetLastError();
        if (hipSuccess != err)
        {
            fprintf(stderr, "getLastCudaError() CUDA error: %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        hipMemcpy(&continue_flag, dev_continue_flag, sizeof(int), hipMemcpyDeviceToHost);
        //hipMemcpy(colors, dev_colors, sizeof(int) * n_vertices, hipMemcpyDeviceToHost);
        //std::cout << "num_finshed=" << num_finished(n_vertices, row_ptr, col, col_ptr, row, colors) << std::endl;
    }
    gettimeofday(&end_time, NULL);
    std::cout << "Main loop time: " << elapsed(start_time, end_time) << std::endl;
    hipMemcpy(colors, dev_colors, sizeof(int) * n_vertices, hipMemcpyDeviceToHost);

    hipFree(dev_continue_flag);
    hipFree(dev_srcs);
    hipFree(dev_dsts);
    hipFree(dev_colors);
    hipFree(dev_locks);
    free(locks);
}

__global__ void mix_color_kernel(const int n_edges, 
                                 const int * srcs, 
                                 const int * dsts, 
                                 int * undone, 
                                 int * colors,
                                 int * continue_flag,
                                int * same_color_sum)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; 
        i < n_edges; i += blockDim.x * gridDim.x)
    {
        int src = srcs[i];
        int dst = dsts[i];
        if (colors[src] == colors[dst])
        {
            colors[dst] = colors[src] + 1;
            undone[dst] = 1;
            //undone[src] = 1;
            *continue_flag = 1;
            //atomicAdd(same_color_sum,1);
        }
    }
}

// color the vertices which are targets
void greedyColor2(const int n_vertices, 
                  const int *row_ptr, 
                  const int *col, 
                  const int * col_ptr, 
                  const int * row, 
                  const int start_c, 
                  const int * is_target,
                  int *colors)
{
  int end_c = start_c; 
  int num_colors = 0;
  bool * color_used = (bool*)malloc(sizeof(bool) * n_vertices);
  memset(color_used, 0, sizeof(bool) * n_vertices);
  for (int i = 0; i < n_vertices; ++i)
  {
    if (is_target[i] == 0)
        continue;
    memset(color_used + start_c, 0, sizeof(bool) * num_colors);
    // Mark the colors used by neighbors
    // Traverse its destinations
    for (int j = row_ptr[i]; j < row_ptr[i+1]; ++j)
    {
      int dst = col[j];
      if (is_target[dst] == 0) continue;

      int c = colors[dst];
      if (c >= start_c)
        color_used[c] = true;
    }

    // Traverse its sources 
    for (int j = col_ptr[i]; j < col_ptr[i+1]; ++j)
    {
      int src = row[j];
      if (is_target[src] == 0) continue;

      int c = colors[src];
      if (c >= start_c)
        color_used[c] = true;
    }

    // select a color which can be used
    for (int j = start_c; j < end_c; ++j)
    {
      if (color_used[j] == false)
        colors[i] = j;
    }
    // add a new color if there is no suitable color
    if (colors[i] < start_c)
    {
      colors[i] = end_c++;
      num_colors++;
    }
  }
}

void mixColor(const int n_vertices, 
              const int n_edges, 
              const int * srcs, 
              const int * dsts, 
              const int *row_ptr, 
              const int *col, 
              const int *col_ptr, 
              const int *row,
              const int niters,
              const float fraction,
              int * colors)
{
    struct timeval start_time, end_time;
    int * dev_srcs;
    int * dev_dsts;
    int * dev_colors;
    int * dev_undone;
    int * dev_continue_flag;
    int continue_flag = 1;
    int * undone;

    gettimeofday(&start_time, NULL);
    undone = (int *)malloc(sizeof(int) * n_vertices);
    memset(colors, 0, sizeof(int) * n_vertices);
    memset(undone, 0, sizeof(int) * n_vertices);
    hipMalloc(&dev_srcs, sizeof(int) * n_edges);
    hipMalloc(&dev_dsts, sizeof(int) * n_edges);
    hipMalloc(&dev_colors, sizeof(int) * n_vertices);
    hipMalloc(&dev_undone, sizeof(int) * n_vertices);
    hipMalloc(&dev_continue_flag, sizeof(int));
    hipMalloc(&same_color_sum,sizeof(int));
    gettimeofday(&end_time, NULL);
    //std::cout << "cuda malloc time: " << elapsed(start_time, end_time) << "ms" << std::endl;

    gettimeofday(&start_time, NULL);
    hipMemcpy(dev_srcs, srcs, sizeof(int) * n_edges, hipMemcpyHostToDevice);
    hipMemcpy(dev_dsts, dsts, sizeof(int) * n_edges, hipMemcpyHostToDevice);
    hipMemcpy(dev_colors, colors, sizeof(int) * n_vertices, hipMemcpyHostToDevice);
    hipMemcpy(dev_undone, undone, sizeof(int) * n_vertices, hipMemcpyHostToDevice);
    gettimeofday(&end_time, NULL);
    //std::cout << "cuda memcpy time: " << elapsed(start_time, end_time) << "ms" << std::endl;

    //gettimeofday(&start_time, NULL);
    int iter = 0;
    float kernel_time = 0;
    int same_color_sum_h;
    int lun_mix = 1;
    while(continue_flag) {
        gettimeofday(&start_time, NULL);
        int nt = 256;
        int nb = (n_vertices + nt - 1)/nt; 
        nb = nb > MAX_GRID_SIZE ? MAX_GRID_SIZE : nb;
        hipMemset(dev_continue_flag, 0, sizeof(int));
        //hipMemset(same_color_sum, 0, sizeof(int));
        hipMemset(dev_undone, 0, sizeof(int) * n_vertices);
        mix_color_kernel<<<nb, nt>>>(n_edges,
                                     dev_srcs, dev_dsts, 
                                     dev_undone, dev_colors,
                                     dev_continue_flag,
                                    same_color_sum);

        hipError_t err = hipGetLastError();
        if (hipSuccess != err)
        {
            fprintf(stderr, "getLastCudaError() CUDA error: %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        //hipMemcpy(&same_color_sum_h, same_color_sum, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&continue_flag, dev_continue_flag, sizeof(int), hipMemcpyDeviceToHost);
        //printf("%d : samecolor_sum = %d\n",lun_mix,same_color_sum_h);
        gettimeofday(&end_time, NULL);
        //lun_mix++;
        float time = elapsed(start_time, end_time);
        kernel_time += time;
        hipMemcpy(undone, dev_undone, sizeof(int) * n_vertices, hipMemcpyDeviceToHost);
        int num_undone = std::count(undone, undone + n_vertices, 1);
        /************
        std::cout << "Iter: " << iter << " "
                  << "num_undone = " << num_undone << " " 
                  << "time = " << time << "ms" << std::endl;
        **************/
        iter++;
        if (niters > 0 && iter >= niters) break;
        if (fraction > 0 && (1 - (float)num_undone / n_vertices) >= fraction) break;
    }
    //gettimeofday(&end_time, NULL);
    //std::cout << "kernel time on gpu: " << elapsed(start_time, end_time) << "ms" << std::endl;

    gettimeofday(&start_time, NULL);
    hipMemcpy(colors, dev_colors, sizeof(int) * n_vertices, hipMemcpyDeviceToHost);
    if (continue_flag)
    {
        hipMemcpy(undone, dev_undone, sizeof(int) * n_vertices, hipMemcpyDeviceToHost);
        /***********************************************************************************
        std::cout << "After coloring on gpu: num_CC = " << getNumCC(n_vertices, row_ptr, col, col_ptr, row, undone) << ", "
                  << "num_undone = " << std::count(undone, undone + n_vertices, 1) << std::endl;
        ***********************************************************************************/
        greedyColor2(n_vertices, row_ptr, col, col_ptr, row, 1000, undone, colors) ;
    }
    gettimeofday(&end_time, NULL);
    float iter_time = 0.0;
    float trav_time = 0.0;
    trav_time = elapsed(start_time, end_time);
    iter_time = kernel_time;
    //std::cout << "Greedy color time for the rest vertices: " << elapsed(start_time, end_time) << "ms" << std::endl;
    //std::cout << "Time(without malloc & memcpy): " << kernel_time + elapsed(start_time, end_time) << "ms" << std::endl;
    std::cout << iter_time << "\t" << trav_time << "\t";



    hipFree(dev_continue_flag);
    hipFree(dev_srcs);
    hipFree(dev_dsts);
    hipFree(dev_colors);
    hipFree(dev_undone);
    free(undone);
}
