#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>

#define SIZE (100*1024*1024)

__global__ void histo_kernel(unsigned char *buffer,long size,unsigned int *histo)
{
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    int tid = blockIdx.x*blockDim.x+threadIdx.x;
    int offset = blockDim.x*gridDim.x;

    while(tid < size)
    {
        atomicAdd(&(temp[buffer[tid]]),1);
        tid+=offset;
    }
    __syncthreads();

    atomicAdd(&(histo[threadIdx.x]),temp[threadIdx.x]);//这里限制了一个线程块只能是256个线程
}

int main(void)
{
    unsigned char *buffer = (unsigned char *)big_random_block(SIZE);//应该是自己写的函数
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    unsigned char *dev_buffer;
    unsigned int *dev_histo;
    hipMalloc((void **)&dev_buffer,SIZE);
    hipMemcpy(dev_buffer,buffer,SIZE,hipMemcpyHostToDevice);
    hipMalloc((void **)&dev_histo,sizeof(int)*256);
    hipMemset(dev_histo,0,256*sizeof(int));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,0);
    int blocks = prop.multiProcessorCount;//确定线程块块数

    histo_kernel<<<blocks*2,256>>>(dev_buffer,SIZE,dev_histo);
    unsigned int histo[256];
    hipMemcpy(histo,dev_histo,sizeof(int)*256,hipMemcpyDeviceToHost);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf("Time to generate: %3.1f ms\n",elapsedTime);//计算用时

    long histoCount = 0;//大致估计正确性
    for(int i=0;i<256;i++)
        histoCount+=histo[i];
    printf("Histogram Sum:%ld\n",histoCount);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(dev_histo);
    hipFree(dev_buffer);
    free(buffer);
    return 0;
}